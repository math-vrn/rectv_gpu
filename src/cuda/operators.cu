#include "hip/hip_runtime.h"

#include "rectv.cuh"
#include "kernels_operators.cuh"
#include <stdio.h>

void rectv::radonapr(float *g, float *f, float tau, int igpu, hipStream_t s)
{
    //tmp arrays on gpus
    float2 *ftmp0 = (float2 *)ftmp[igpu];
    float2 *ftmps0 = (float2 *)ftmps[igpu];
    float2 *gtmp0 = (float2 *)gtmp[igpu];
    float2 *phi0 = (float2 *)phi[igpu];
    float *theta0 = (float *)theta[igpu];

    hipMemsetAsync(ftmp0, 0, 2 * n * n * m * nzp * sizeof(float), s);
    hipMemsetAsync(ftmps0, 0, 2 * n * n * nzp * sizeof(float), s);
    hipMemsetAsync(gtmp0, 0, 2 * n * ntheta * nzp * sizeof(float), s);
    hipMemsetAsync(g, 0, n * ntheta * nzp * sizeof(float), s);

    //switch to complex numbers
    makecomplex<<<GS3d0, BS3d, 0, s>>>(ftmp0, f, n, n, m * nzp);
    for (int i = 0; i < m; i++)
    {
        //decompositon coefficients
        decphi<<<GS3d1, BS3d, 0, s>>>(ftmps0, ftmp0, &phi0[i * ntheta], n, ntheta, m, nzp);
        rad[igpu]->fwdR(gtmp0, ftmps0, theta0, s);
        //multiplication by basis functions
        mulphi<<<GS3d2, BS3d, 0, s>>>(gtmp0, &phi0[i * ntheta], 1, m, n, ntheta, nzp);
        //sum up
        addreal<<<GS3d2, BS3d, 0, s>>>(g, gtmp0, tau, n, ntheta, nzp);
    }
}

void rectv::radonapradj(float *f, float *g, float tau, int igpu, hipStream_t s)
{
    //tmp arrays on gpus
    float2 *ftmp0 = (float2 *)ftmp[igpu];
    float2 *ftmps0 = (float2 *)ftmps[igpu];
    float2 *gtmp0 = (float2 *)gtmp[igpu];
    float2 *phi0 = (float2 *)phi[igpu];
    float *theta0 = (float *)theta[igpu];

    hipMemsetAsync(ftmp0, 0, 2 * n * n * m * nzp * sizeof(float), s);
    hipMemsetAsync(ftmps0, 0, 2 * n * n * nzp * sizeof(float), s);
    hipMemsetAsync(gtmp0, 0, 2 * n * ntheta * nzp * sizeof(float), s);

    for (int i = 0; i < m; i++)
    {
        //switch to complex numbers
        makecomplex<<<GS3d2, BS3d, 0, s>>>(gtmp0, g, n, ntheta, nzp);
        //multiplication by conjugate basis functions
        mulphi<<<GS3d2, BS3d, 0, s>>>(gtmp0, &phi0[i * ntheta], -1, m, n, ntheta, nzp); //-1 conj       
        rad[igpu]->adjR(ftmps0, gtmp0, theta0, 0, s);
        //recovering by coefficients
        recphi<<<GS3d1, BS3d, 0, s>>>(ftmp0, ftmps0, &phi0[i * ntheta], n, ntheta, m, nzp);
    }
    addreal<<<GS3d0, BS3d, 0, s>>>(f, ftmp0, tau, n, n, m * nzp);
}

void rectv::gradient(float4 *h2, float *f, float lambda1, int iz, int igpu, hipStream_t s)
{
    float *ftmp0 = ftmp[igpu];
    //repeat border values
    extendf<<<GS3d3, BS3d, 0, s>>>(ftmp0, f, iz != 0, iz != nz / nzp - 1, n + 2, m + 2, nzp + 2);
    gradf<<<GS3d3, BS3d, 0, s>>>(h2, ftmp0, lambda1, n + 1, m + 1, nzp + 1);
}

void rectv::divergent(float *fn, float4 *h2, float lambda1, float tau, int igpu, hipStream_t s)
{
    div<<<GS3d0, BS3d, 0, s>>>(fn, h2, tau, lambda1, n, m, nzp);
}
