#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include "rectv.cuh"

rectv::rectv(size_t N_, size_t Ntheta_, size_t M_, size_t Nz_, size_t Nzp_, size_t ngpus_, float center, float lambda0_, float lambda1_)
{
	N = N_;
	Ntheta = Ntheta_;
	M = M_;
	Nz = Nz_;
	Nzp = Nzp_;
	lambda0 = lambda0_;
	lambda1 = lambda1_;
	ngpus = min(ngpus_, (size_t)(Nz / Nzp));
	omp_set_num_threads(ngpus);
	//Managed memory on GPU
	hipMallocManaged((void **)&f, N * N * M * Nz * sizeof(float));
	hipMallocManaged((void **)&fn, N * N * M * Nz * sizeof(float));
	hipMallocManaged((void **)&ft, N * N * M * Nz * sizeof(float));
	hipMallocManaged((void **)&ftn, N * N * M * Nz * sizeof(float));
	hipMallocManaged((void **)&g, N * Ntheta * Nz * sizeof(float));
	hipMallocManaged((void **)&h1, N * Ntheta * Nz * sizeof(float));
	hipMallocManaged((void **)&h2, (N + 1) * (N + 1) * (M + 1) * (Nzp + 1) * Nz / Nzp * sizeof(float4));

	//Class for applying Radon transform
	rad = new radonusfft *[ngpus];
	//tmp arrays
	ftmp = new float *[ngpus];
	gtmp = new float *[ngpus];
	ftmps = new float *[ngpus];
	phi = new float2 *[ngpus];
	theta = new float *[ngpus];

	BS2d = dim3(32, 32);
	BS3d = dim3(32, 32, 1);
	GS2d0 = dim3(ceil(Ntheta / (float)BS2d.x), ceil(M / (float)BS2d.y));

	GS3d0 = dim3(ceil(N / (float)BS3d.x), ceil(N / (float)BS3d.y), ceil(Nzp * M / (float)BS3d.z));
	GS3d1 = dim3(ceil(N / (float)BS3d.x), ceil(N / (float)BS3d.y), ceil(Nzp / (float)BS3d.z));
	GS3d2 = dim3(ceil(N / (float)BS3d.x), ceil(Ntheta / (float)BS3d.y), ceil(Nzp / (float)BS3d.z));
	GS3d3 = dim3(ceil((N + 2) / (float)BS3d.x), ceil((N + 2) / (float)BS3d.y), ceil((M + 2) * (Nzp + 2) / (float)BS3d.z));
	GS3d4 = dim3(ceil((N + 1) / (float)BS3d.x), ceil((N + 1) / (float)BS3d.y), ceil((M + 1) * (Nzp + 1) / (float)BS3d.z));

	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		hipSetDevice(igpu);
		rad[igpu] = new radonusfft(N, Ntheta, Nzp, center);
		hipMalloc((void **)&ftmp[igpu], 2 * (N + 2) * (N + 2) * (M + 2) * (Nzp + 2) * sizeof(float));
		hipMalloc((void **)&gtmp[igpu], 2 * N * Ntheta * Nzp * sizeof(float));
		hipMalloc((void **)&ftmps[igpu], 2 * N * N * Nzp * sizeof(float));
		hipMalloc((void **)&phi[igpu], 2 * Ntheta * M * sizeof(float));
		hipMalloc((void **)&theta[igpu], Ntheta * sizeof(float));		
	}
	hipDeviceSynchronize();
}

rectv::~rectv()
{
	hipFree(f);
	hipFree(fn);
	hipFree(ft);
	hipFree(ftn);
	hipFree(g);
	hipFree(h1);
	hipFree(h2);
	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		hipSetDevice(igpu);
		delete rad[igpu];
		hipFree(ftmp[igpu]);
		hipFree(gtmp[igpu]);
		hipFree(ftmps[igpu]);
		hipFree(phi[igpu]);
		hipFree(theta[igpu]);
		hipDeviceReset();
	}
}

void rectv::run(float *fres, float *g_, float *theta_, float *phi_, size_t niter)
{
	//data
	hipMemcpy(g, g_, N * Ntheta * Nz * sizeof(float), hipMemcpyHostToHost);
	//angles and basis functions to each gpu
	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		hipSetDevice(igpu);
		hipMemcpy(theta[igpu], theta_, Ntheta * sizeof(float), hipMemcpyDefault);
		hipMemcpy(phi[igpu], phi_, 2 * Ntheta * M * sizeof(float), hipMemcpyDefault);
	}
	//initial guess
	memset(f, 0, N * N * M * Nz * sizeof(float));
	memset(ft, 0, N * N * M * Nz * sizeof(float));
	memset(fn, 0, N * N * M * Nz * sizeof(float));
	memset(ftn, 0, N * N * M * Nz * sizeof(float));
	memset(h1, 0, N * Ntheta * Nz * sizeof(float));
	memset(h2, 0, (N + 1) * (N + 1) * (M + 1) * (Nzp + 1) * Nz / Nzp * sizeof(float4));

	float start = omp_get_wtime();
#pragma omp parallel
	{
		int igpu = omp_get_thread_num();
		hipSetDevice(igpu);
		hipStream_t s1, s2, s3, st;
		hipEvent_t e1, e2, et;
		hipStreamCreate(&s1);
		hipStreamCreate(&s2);
		hipStreamCreate(&s3);
		hipEventCreate(&e1);
		hipEventCreate(&e2);
		for (int iter = 0; iter < niter; iter++)
		{
			//parts in z
			int iz = igpu * Nz / Nzp / ngpus;
			float *f0 = &f[N * N * M * iz * Nzp];
			float *fn0 = &fn[N * N * M * iz * Nzp];
			float *ft0 = &ft[N * N * M * iz * Nzp];
			float *ftn0 = &ftn[N * N * M * iz * Nzp];
			float *h10 = &h1[N * Ntheta * iz * Nzp];
			float4 *h20 = &h2[(N + 1) * (N + 1) * (M + 1) * iz * (Nzp + 1)];
			float *g0 = &g[N * Ntheta * iz * Nzp];
			hipMemPrefetchAsync(f0, N * N * M * Nzp * sizeof(float), igpu, s2);																	//mem+=N*N*M*Nzp*sizeof(float);
			hipMemPrefetchAsync(fn0, N * N * M * Nzp * sizeof(float), igpu, s2);																	//mem+=N*N*M*Nzp*sizeof(float);
			hipMemPrefetchAsync(&ft0[-(iz != 0) * N * N * M], N * N * M * (Nzp + 2 - (iz == 0) - (iz == Nz / Nzp - 1)) * sizeof(float), igpu, s2); //mem+=N*N*M*(Nzp+2-(iz==0)-(iz==Nz/Nzp-1))*sizeof(float);
			hipMemPrefetchAsync(ftn0, N * N * M * Nzp * sizeof(float), igpu, s2);																	//mem+=N*N*M*Nzp*sizeof(float);
			hipMemPrefetchAsync(h10, N * Ntheta * Nzp * sizeof(float), igpu, s2);																	//mem+=N*Ntheta*Nzp*sizeof(float);
			hipMemPrefetchAsync(h20, (N + 1) * (N + 1) * (M + 1) * (Nzp + 1) * sizeof(float4), igpu, s2);											//mem+=(N+1)*(N+1)*(M+1)*(Nzp+1)*sizeof(float4);
			hipMemPrefetchAsync(g0, N * Ntheta * Nzp * sizeof(float), igpu, s2);																	//mem+= N*Ntheta*Nzp*sizeof(float);

			hipEventRecord(e1, s2);
			float *f0s = f0;
			float *fn0s = fn0;
			float *ft0s = ft0;
			float *ftn0s = ftn0;
			float *h10s = h10;
			float4 *h20s = h20;
			float *g0s = g0;
#pragma omp forrectv
			for (int iz = 0; iz < Nz / Nzp; iz++)
			{
				hipEventSynchronize(e1);
				hipEventSynchronize(e2);

				solver_chambolle(f0, fn0, ft0, ftn0, h10, h20, g0, iz, igpu, s1);

				hipEventRecord(e1, s1);
				if (iz < (igpu + 1) * Nz / Nzp / ngpus - 1)
				{
					// make sure the stream is idle to force non-deferred HtoD prefetches first
					hipStreamSynchronize(s2);
					//parts in z
					f0s = &f[N * N * M * (iz + 1) * Nzp];
					fn0s = &fn[N * N * M * (iz + 1) * Nzp];
					ft0s = &ft[N * N * M * (iz + 1) * Nzp];
					ftn0s = &ftn[N * N * M * (iz + 1) * Nzp];
					h10s = &h1[N * Ntheta * (iz + 1) * Nzp];
					h20s = &h2[(N + 1) * (N + 1) * (M + 1) * (iz + 1) * (Nzp + 1)];
					g0s = &g[N * Ntheta * (iz + 1) * Nzp];
					hipMemPrefetchAsync(f0s, N * N * M * Nzp * sizeof(float), igpu, s2);											//mem+=N*N*M*Nzp*sizeof(float);
					hipMemPrefetchAsync(fn0s, N * N * M * Nzp * sizeof(float), igpu, s2);											//mem+=N*N*M*Nzp*sizeof(float);
					hipMemPrefetchAsync(&ft0s[N * N * M], N * N * M * (Nzp - (iz + 1 == Nz / Nzp - 1)) * sizeof(float), igpu, s2); //mem+=N*N*M*(Nzp-(iz+1==Nz/Nzp-1))*sizeof(float);
					hipMemPrefetchAsync(ftn0s, N * N * M * Nzp * sizeof(float), igpu, s2);											//mem+=N*N*M*Nzp*sizeof(float);
					hipMemPrefetchAsync(h10s, N * Ntheta * Nzp * sizeof(float), igpu, s2);											//mem+=N*Ntheta*Nzp*sizeof(float);
					hipMemPrefetchAsync(h20s, (N + 1) * (N + 1) * (M + 1) * (Nzp + 1) * sizeof(float4), igpu, s2);					//mem+=(N+1)*(N+1)*(M+1)*(Nzp+1)*sizeof(float4);
					hipMemPrefetchAsync(g0s, N * Ntheta * Nzp * sizeof(float), igpu, s2);											//mem+=N*Ntheta*Nzp*sizeof(float);

					hipEventRecord(e2, s2);
				}

				hipMemPrefetchAsync(f0, N * N * M * Nzp * sizeof(float), hipCpuDeviceId, s1);																												   //mem+=N*N*M*Nzp*sizeof(float);
				hipMemPrefetchAsync(fn0, N * N * M * Nzp * sizeof(float), hipCpuDeviceId, s1);																											   //mem+=N*N*M*Nzp*sizeof(float);
				hipMemPrefetchAsync(&ft0[-(iz != 0) * N * N * M], N * N * M * (Nzp - (iz == 0) - (iz == Nz / Nzp - 1) + 2 * (iz == (igpu + 1) * Nz / Nzp / ngpus - 1)) * sizeof(float), hipCpuDeviceId, s1); //mem+= N*N*M*(Nzp-(iz==0)-(iz==Nz/Nzp-1)+2*(iz==(igpu+1)*Nz/Nzp/ngpus-1))*sizeof(float);

				hipMemPrefetchAsync(ftn0, N * N * M * Nzp * sizeof(float), hipCpuDeviceId, s1);						  //mem+=N*N*M*Nzp*sizeof(float);
				hipMemPrefetchAsync(h10, N * Ntheta * Nzp * sizeof(float), hipCpuDeviceId, s1);						  //mem+=N*Ntheta*Nzp*sizeof(float);
				hipMemPrefetchAsync(h20, (N + 1) * (N + 1) * (M + 1) * (Nzp + 1) * sizeof(float4), hipCpuDeviceId, s1); //mem+=(N+1)*(N+1)*(M+1)*(Nzp+1)*sizeof(float4);
				hipMemPrefetchAsync(g0, N * Ntheta * Nzp * sizeof(float), hipCpuDeviceId, s1);						  //mem+=N*Ntheta*Nzp*sizeof(float);

				f0 = f0s;
				fn0 = fn0s;
				ft0 = ft0s;
				ftn0 = ftn0s;
				h10 = h10s;
				h20 = h20s;
				g0 = g0s;
				// rotate streams and swap events
				st = s1;
				s1 = s2;
				s2 = st;
				st = s2;
				s2 = s3;
				s3 = st;
				et = e1;
				e1 = e2;
				e2 = et;
			}

			hipEventSynchronize(e1);
			hipEventSynchronize(e2);
			hipDeviceSynchronize();
#pragma omp barrier
#pragma omp single
			{
				float *tmp = 0;
				tmp = ft;
				ft = ftn;
				ftn = tmp;
				tmp = f;
				f = fn;
				fn = tmp;

				float norm[2] = {};
				for (int k = 0; k < N * Ntheta * Nz; k++)
					norm[0] += (h1[k] - g[k]) * (h1[k] - g[k]);
				for (int k = 0; k < (N + 1) * (N + 1) * (M + 1) * (Nzp + 1) * Nz / Nzp; k++)
					norm[1] += sqrt(h2[k].x * h2[k].x + h2[k].y * h2[k].y + h2[k].z * h2[k].z + h2[k].w * h2[k].w);
				fprintf(stderr, "iterations (%d/%d) f:%f, r:%f, total:%f\n", iter, niter, norm[0], lambda0 * norm[1], norm[0] + lambda0 * norm[1]);
				fflush(stdout);
			}
		}
		hipDeviceSynchronize();
#pragma omp barrier
	}
	float end = omp_get_wtime();
	printf("Elapsed time: %fs.\n", end - start);
	hipMemPrefetchAsync(ft, N * N * M * Nz * sizeof(float), hipCpuDeviceId, 0);
	hipMemcpy(fres, ft, N * N * M * Nz * sizeof(float), hipMemcpyDefault);
}

void rectv::run_wrap(float *fres, int N0, float *g, int N1, float *theta, int N2, float *phi, int N3, size_t niter)
{
	run(fres, g, theta, phi, niter);
}