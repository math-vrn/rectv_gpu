#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include "rectv.cuh"

rectv::rectv(int n, int ntheta, int m, int nz, int nzp_, int ngpus_)
 : n(n), ntheta(ntheta), m(m), nz(nz) {

	nzp = nzp_;
	ngpus = min(ngpus_, (int)(nz / nzp));
	omp_set_num_threads(ngpus);
	//Managed memory on GPU
	hipMallocManaged((void **)&f, n * n * m * nz * sizeof(float));
	hipMallocManaged((void **)&fn, n * n * m * nz * sizeof(float));
	hipMallocManaged((void **)&g, n * ntheta * nz * sizeof(float));
	hipMallocManaged((void **)&psi, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * nz / nzp * sizeof(float4));
	hipMallocManaged((void **)&mu, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * nz / nzp * sizeof(float4));

	//Class for applying Radon transform
	rad = new radonusfft *[ngpus];
	cublas_handles = new hipblasHandle_t[ngpus];
	//tmp arrays
	ftmp = new float *[ngpus];
	gtmp = new float *[ngpus];
	ftmps = new float *[ngpus];
	fm = new float *[ngpus];
	h1 = new float *[ngpus];
	h2 = new float4 *[ngpus];
	h2stored = new float4 *[ngpus];
	phi = new float2 *[ngpus];
	theta = new float *[ngpus];
	
	BS2d = dim3(32, 32);
	BS3d = dim3(32, 32, 1);
	GS2d0 = dim3(ceil(ntheta / (float)BS2d.x), ceil(m / (float)BS2d.y));

	GS3d0 = dim3(ceil(n / (float)BS3d.x), ceil(n / (float)BS3d.y), ceil(nzp * m / (float)BS3d.z));
	GS3d1 = dim3(ceil(n / (float)BS3d.x), ceil(n / (float)BS3d.y), ceil(nzp / (float)BS3d.z));
	GS3d2 = dim3(ceil(n / (float)BS3d.x), ceil(ntheta / (float)BS3d.y), ceil(nzp / (float)BS3d.z));
	GS3d3 = dim3(ceil((n + 2) / (float)BS3d.x), ceil((n + 2) / (float)BS3d.y), ceil((m + 2) * (nzp + 2) / (float)BS3d.z));
	GS3d4 = dim3(ceil((n + 1) / (float)BS3d.x), ceil((n + 1) / (float)BS3d.y), ceil((m + 1) * (nzp + 1) / (float)BS3d.z));

	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		hipSetDevice(igpu);
		rad[igpu] = new radonusfft(n, ntheta, nzp);
		hipMalloc((void **)&ftmp[igpu], 2 * (n + 2) * (n + 2) * (m + 2) * (nzp + 2) * sizeof(float));
		hipMalloc((void **)&gtmp[igpu], 2 * n * ntheta * nzp * sizeof(float));
		hipMalloc((void **)&ftmps[igpu], 2 * n * n * nzp * sizeof(float));
		hipMalloc((void **)&h1[igpu], n * ntheta * nzp * sizeof(float));
		hipMalloc((void **)&h2[igpu], (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * sizeof(float4));		
		hipMalloc((void **)&h2stored[igpu], (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * sizeof(float4));				
		hipMalloc((void **)&phi[igpu], 2 * ntheta * m * sizeof(float));
		hipMalloc((void **)&fm[igpu], n * n * m * (nzp + 2) * sizeof(float));
		hipMalloc((void **)&theta[igpu], ntheta * sizeof(float));		
		hipblasCreate(&cublas_handles[igpu]);
	}
	hipDeviceSynchronize();
	is_free = false;
}

rectv::~rectv() { free(); }

void rectv::free()
{
	if (!is_free) 
	{
		hipFree(f);
		hipFree(fn);
		hipFree(fm);
		hipFree(g);
		hipFree(psi);
		hipFree(mu);
		for (int igpu = 0; igpu < ngpus; igpu++)
		{
			hipSetDevice(igpu);
			delete rad[igpu];
			hipFree(ftmp[igpu]);
			hipFree(gtmp[igpu]);
			hipFree(ftmps[igpu]);
			hipFree(fm[igpu]);
			hipFree(h1[igpu]);
			hipFree(h2[igpu]);		
			hipFree(h2stored[igpu]);		
			hipFree(phi[igpu]);
			hipFree(theta[igpu]);
			hipblasDestroy(cublas_handles[igpu]);
			hipDeviceReset();
		}
		is_free = true;
	}
}

void rectv::set_center(float center)
{
	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		hipSetDevice(igpu);
		rad[igpu]->set_center(center);
	}
}

void rectv::run(size_t fres, size_t g_, size_t theta_, size_t phi_, 
	float center, float lambda0, float lambda1, 
	int niter, int titer)
{
	//update center with a given one
	set_center(center);
	
	//copy data
	hipMemcpy(g, (float*)g_, n * ntheta * nz * sizeof(float), hipMemcpyHostToHost);
	//angles and basis functions to each gpu
	for (int igpu = 0; igpu < ngpus; igpu++)
	{
		hipSetDevice(igpu);
		hipMemcpy(theta[igpu], (float*)theta_, ntheta * sizeof(float), hipMemcpyDefault);
		hipMemcpy(phi[igpu], (float*)phi_, 2 * ntheta * m * sizeof(float), hipMemcpyDefault);
	}
	//initial guess
	memset(f, 0, n * n * m * nz * sizeof(float));
	memset(fn, 0, n * n * m * nz * sizeof(float));
	memset(psi, 0, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * nz / nzp * sizeof(float4));
	memset(mu, 0, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * nz / nzp * sizeof(float4));

	//weighting factor for the ADMM scheme
	float rho = 0.5;
	//norms for updating the weighting factor on each iteration
	float2* normdiff = new float2[nz/nzp];
#pragma omp parallel
	{
		int igpu = omp_get_thread_num();
	
		hipSetDevice(igpu);
		hipStream_t s1, s2, s3, st;
		hipEvent_t e1, e2, et;
		hipStreamCreate(&s1);
		hipStreamCreate(&s2);
		hipStreamCreate(&s3);
		hipEventCreate(&e1);
		hipEventCreate(&e2);
		for (int iter = 0; iter < niter; iter++)
		{
			//parts in z
			int iz = igpu * nz / nzp / ngpus;
			float *f0 = &f[n * n * m * iz * nzp];
			float *fn0 = &fn[n * n * m * iz * nzp];
			float4 *psi0 = &psi[(n + 1) * (n + 1) * (m + 1) * iz * (nzp + 1)];
			float4 *mu0 = &mu[(n + 1) * (n + 1) * (m + 1) * iz * (nzp + 1)];
			float *g0 = &g[n * ntheta * iz * nzp];
			hipMemPrefetchAsync(&f0[-(iz != 0) * n * n * m], n * n * m * (nzp + 2 - (iz == 0) - (iz == nz / nzp - 1)) * sizeof(float), igpu, s2); //mem+=n*n*m*(nzp+2-(iz==0)-(iz==nz/nzp-1))*sizeof(float);
			hipMemPrefetchAsync(fn0, n * n * m * nzp * sizeof(float), igpu, s2);																	//mem+=n*n*m*nzp*sizeof(float);
			hipMemPrefetchAsync(psi0, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * sizeof(float4), igpu, s2);											//mem+=(n+1)*(n+1)*(m+1)*(nzp+1)*sizeof(float4);
			hipMemPrefetchAsync(mu0, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * sizeof(float4), igpu, s2);											//mem+=(n+1)*(n+1)*(m+1)*(nzp+1)*sizeof(float4);
			hipMemPrefetchAsync(g0, n * ntheta * nzp * sizeof(float), igpu, s2);																	//mem+= n*ntheta*nzp*sizeof(float);
			
			hipEventRecord(e1, s2);
			float *f0s = f0;
			float *fn0s = fn0;
			float4 *psi0s = psi0;
			float4 *mu0s = mu0;
			float *g0s = g0;
#pragma omp for
			for (int iz = 0; iz < nz / nzp; iz++)
			{
				hipEventSynchronize(e1);
				hipEventSynchronize(e2);
				// intermediate arrays
				float* h10 = h1[igpu];
    			float4* h20 = h2[igpu];
				float4* h2stored0 = h2stored[igpu];
				float* fm0 = &fm[igpu][(iz != 0) * n * n * m];//modifyable version of f
    			hipMemcpyAsync(&fm0[-(iz != 0) * n * n * m],&f0[-(iz != 0) * n * n * m],n*n*(nzp + 2 - (iz == 0) - (iz == nz / nzp - 1))*m* sizeof(float), hipMemcpyDefault, s1); //mem+=n*n*m*(nzp+2-(iz==0)-(iz==nz/nzp-1))*sizeof(float);
				// ADMM
				normdiff[iz] = solver_admm(f0, fn0, h10, h20, h2stored0, fm0, g0, psi0, mu0, lambda0, lambda1, rho, iz, titer, igpu, s1);

				hipEventRecord(e1, s1);
				if (iz < (igpu + 1) * nz / nzp / ngpus - 1)
				{
					// make sure the stream is idle to force non-deferred HtoD prefetches first
					hipStreamSynchronize(s2);
					//parts in z
					f0s = &f[n * n * m * (iz + 1) * nzp];
					fn0s = &fn[n * n * m * (iz + 1) * nzp];
					psi0s = &psi[(n + 1) * (n + 1) * (m + 1) * (iz + 1) * (nzp + 1)];
					mu0s = &mu[(n + 1) * (n + 1) * (m + 1) * (iz + 1) * (nzp + 1)];
					g0s = &g[n * ntheta * (iz + 1) * nzp];
					hipMemPrefetchAsync(&f0s[n * n * m], n * n * m * (nzp - (iz + 1 == nz / nzp - 1)) * sizeof(float), igpu, s2); //mem+=n*n*m*(nzp-(iz+1==nz/nzp-1))*sizeof(float);
					hipMemPrefetchAsync(fn0s, n * n * m * nzp * sizeof(float), igpu, s2);											//mem+=n*n*m*nzp*sizeof(float);
					hipMemPrefetchAsync(psi0s, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * sizeof(float4), igpu, s2);					//mem+=(n+1)*(n+1)*(m+1)*(nzp+1)*sizeof(float4);
					hipMemPrefetchAsync(mu0s, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * sizeof(float4), igpu, s2);					//mem+=(n+1)*(n+1)*(m+1)*(nzp+1)*sizeof(float4);
					hipMemPrefetchAsync(g0s, n * ntheta * nzp * sizeof(float), igpu, s2);											//mem+=n*ntheta*nzp*sizeof(float);

					hipEventRecord(e2, s2);
				}

				hipMemPrefetchAsync(&f0[-(iz != 0) * n * n * m], n * n * m * (nzp - (iz == 0) - (iz == nz / nzp - 1) + 2 * (iz == (igpu + 1) * nz / nzp / ngpus - 1)) * sizeof(float), hipCpuDeviceId, s1); //mem+= n*n*m*(nzp-(iz==0)-(iz==nz/nzp-1)+2*(iz==(igpu+1)*nz/nzp/ngpus-1))*sizeof(float);
				hipMemPrefetchAsync(fn0, n * n * m * nzp * sizeof(float), hipCpuDeviceId, s1);						  //mem+=n*n*m*nzp*sizeof(float);
				hipMemPrefetchAsync(g0, n * ntheta * nzp * sizeof(float), hipCpuDeviceId, s1);						  //mem+=n*ntheta*nzp*sizeof(float);

				f0 = f0s;
				fn0 = fn0s;
				psi0 = psi0s;
				mu0 = mu0s;
				g0 = g0s;
				// rotate streams and swap events
				st = s1;
				s1 = s2;
				s2 = st;
				st = s2;
				s2 = s3;
				s3 = st;
				et = e1;
				e1 = e2;
				e2 = et;
			}

			hipEventSynchronize(e1);
			hipEventSynchronize(e2);
			hipDeviceSynchronize();
#pragma omp barrier
#pragma omp single
			{

				//update rho
				float r=0;
				float s=0;
				for(int k=0;k<nz/nzp;k++)
				{
					r += normdiff[k].x*normdiff[k].x;
					s += rho*rho*normdiff[k].y*normdiff[k].y;
				}
				if(r>10*s) rho*=2;
				else if (s>10*r) rho*=0.5;
				
				//check convergence
				double norm=0;
				for (int k = 0; k < n * n * m * nz; k++)
					norm += (fn[k] - f[k]) * (fn[k] - f[k]);
				fprintf(stderr, "iter (%d/%d), rdiff ||f(k+1)-f(k)||:%f \n", iter, niter, norm);fflush(stdout);
				
				//swap old/new object
				float *tmp = 0;				
				tmp = f;
				f = fn;
				fn = tmp;			
			}
		}
		hipDeviceSynchronize();
#pragma omp barrier
	}
	hipMemPrefetchAsync(f, n * n * m * nz * sizeof(float), hipCpuDeviceId, 0);
	hipMemcpy((float*)fres, f, n * n * m * nz * sizeof(float), hipMemcpyDefault);
}


void rectv::check_adjoints(size_t res_, size_t g_, size_t theta_, size_t phi_, float lambda1, float center)
{
	double* res = (double*)res_;
    // only for 1 gpu and 1 slice set    
    //update center with a given one
	set_center(center);
    // Rapr
    hipMemcpy(g, (float*)g_, n * ntheta * nz * sizeof(float), hipMemcpyDefault);
    hipMemcpy(theta[0], (float*)theta_, ntheta * sizeof(float), hipMemcpyDefault);
    hipMemcpy(phi[0], (float*)phi_, 2 * ntheta * m * sizeof(float), hipMemcpyDefault);

    radonapradj(f, g, 1, 0, 0);
    radonapr(h1[0], f, 1, 0, 0);
    
    float *ftmp = new float[n * n * nz * m];
    float *h1tmp = new float[n * ntheta * nz];
    hipMemcpy(ftmp, f, n * n * nz * m * sizeof(float), hipMemcpyDefault);    
    hipMemcpy(h1tmp, h1[0], n * ntheta * nz * sizeof(float), hipMemcpyDefault);
    for (int k = 0; k < n * n * nz * m; k++) res[0] += ftmp[k] * ftmp[k];
    for (int k = 0; k < n * ntheta * nz; k++) res[1] += ((float*)g_)[k] * h1tmp[k];
    for (int k = 0; k < n * ntheta * nz; k++) res[2] += h1tmp[k] * h1tmp[k];
    
    // gradient
    gradient(h2[0], f, lambda1, 0, 0, 0);
   
    divergent(fn, h2[0], lambda1, 1,  0, 0);
     
    float *fntmp = new float[n * n * nz * m];
    float *h2tmp = new float[(n + 1) * (n + 1) * (m + 1) * (nz + 1)  * sizeof(float4)];
    hipMemcpy(fntmp, fn, n * n * nz * m * sizeof(float), hipMemcpyDefault);    
    hipMemcpy(h2tmp, h2[0], (n + 1) * (n + 1) * (m + 1) * (nz + 1) *  sizeof(float4), hipMemcpyDefault);
    for (int k = 0; k < n * n * nz * m; k++) res[3] += ftmp[k] * fntmp[k];
    for (int k = 0; k < (n + 1) * (n + 1) * (m + 1) * (nz + 1); k++) res[4] += h2tmp[k] * h2tmp[k];
    for (int k = 0; k < n * n * nz * m; k++) res[5] += fntmp[k] * fntmp[k];
    
}