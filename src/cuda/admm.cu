#include "hip/hip_runtime.h"

#include "rectv.cuh"
#include "stdio.h"

__global__ void diff(float *h1, float *g, int n, int ntheta, int nz)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tz = blockIdx.z * blockDim.z + threadIdx.z;
    if (tx >= n || ty >= ntheta || tz >= nz)
        return;

    int id0 = tx + ty * n + tz * n * ntheta;
    h1[id0] = (h1[id0] - g[id0]);
}

__global__ void updatemu_ker(float4 *mu, float4 *h2, float4 *psi, float rho, int n, int nz)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tz = blockIdx.z * blockDim.z + threadIdx.z;
    if (tx >= n || ty >= n || tz >= nz)
        return;

    int id0 = tx + ty * n + tz * n * n;
    mu[id0].x += rho * (h2[id0].x - psi[id0].x);
    mu[id0].y += rho * (h2[id0].y - psi[id0].y);
    mu[id0].z += rho * (h2[id0].z - psi[id0].z);
    mu[id0].w += rho * (h2[id0].w - psi[id0].w);
}

__global__ void diffgrad(float4 *h2, float4 *psi, float4 *mu, float rho, int n, int nz)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tz = blockIdx.z * blockDim.z + threadIdx.z;
    if (tx >= n || ty >= n || tz >= nz)
        return;

    int id0 = tx + ty * n + tz * n * n;
    h2[id0].x = rho * (h2[id0].x - psi[id0].x + mu[id0].x / rho);
    h2[id0].y = rho * (h2[id0].y - psi[id0].y + mu[id0].y / rho);
    h2[id0].z = rho * (h2[id0].z - psi[id0].z + mu[id0].z / rho);
    h2[id0].w = rho * (h2[id0].w - psi[id0].w + mu[id0].w / rho);
}

__global__ void solve_reg_ker(float4* psi, float4 *h2, float4* mu, float lambda, float rho, int n, int nz)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tz = blockIdx.z * blockDim.z + threadIdx.z;
    if (tx >= n || ty >= n || tz >= nz)
        return;

    int id0 = tx + ty * n + tz * n * n;
    psi[id0].x =(h2[id0].x+mu[id0].x/rho);
    psi[id0].y =(h2[id0].y+mu[id0].y/rho);
    psi[id0].z =(h2[id0].z+mu[id0].z/rho);
    psi[id0].w =(h2[id0].w+mu[id0].w/rho);    
	float za = sqrtf(psi[id0].x * psi[id0].x + 
                     psi[id0].y * psi[id0].y + 
                     psi[id0].z * psi[id0].z + 
                     psi[id0].w * psi[id0].w);
	if (za <= lambda / rho)
	{
		psi[id0].x = 0;
		psi[id0].y = 0;
		psi[id0].z = 0;
		psi[id0].w = 0;
	}
	else
	{
      	psi[id0].x -= lambda / rho * psi[id0].x / za;
	 	psi[id0].y -= lambda / rho * psi[id0].y / za;
		psi[id0].z -= lambda / rho * psi[id0].z / za;
		psi[id0].w -= lambda / rho * psi[id0].w / za;
	}
}

void rectv::cg(float *ft0, float *ftn0, float *h10, float4 *h20, float *g0, float4 *psi0, float4 *mu0, float rho, int iz, int niter, int igpu, hipStream_t s)       
{
    float* ft00 = &fe[igpu][(iz != 0) * n * n * m];//modifyable version of ft0
    hipMemcpyAsync(&ft00[-(iz != 0) * n * n * m],&ft0[-(iz != 0) * n * n * m],n*n*(nzp + 2 - (iz == 0) - (iz == nz / nzp - 1))*m* sizeof(float), hipMemcpyDefault, s); //mem+=n*n*m*(nzp+2-(iz==0)-(iz==nz/nzp-1))*sizeof(float);
    
    for (int k=0;k<niter;k++)
    {
        hipMemsetAsync(h10, 0, n * ntheta * nzp * sizeof(float), s);
        hipMemsetAsync(h20, 0, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * sizeof(float4), s);        
        //forward step
        gradient(h20, ft00, 1, iz, igpu, s); //iz for border control
        radonapr(h10, ft00, 1, igpu, s);        
        //differences
        diffgrad<<<GS3d4, BS3d, 0, s>>>(h20, psi0, mu0, rho, (n + 1), (m + 1) * (nzp + 1));
        diff<<<GS3d2, BS3d, 0, s>>>(h10, g0, n, ntheta, nzp);
        divergent(ft00, ft00, h20, 0.5/lambda1, igpu, s);
        //backward step
        radonapradj(ft00, h10, 0.5/lambda1, igpu, s);         
    }    
    hipMemsetAsync(h20, 0, (n + 1) * (n + 1) * (m + 1) * (nzp + 1) * sizeof(float4), s);        
    //forward step
    gradient(h20, ft00, 1, iz, igpu, s); //iz for border control
    hipMemcpyAsync(ftn0, ft00, n * n * nzp * m * sizeof(float), hipMemcpyDefault, s);
}

void rectv::solve_reg(float4* psi, float4* h2, float4* mu, float lambda, float rho, hipStream_t s)
{
    solve_reg_ker<<<GS3d4, BS3d, 0, s>>>(psi, h2, mu, lambda, rho, n+1, (m+1)*(nzp+1));   
}

void rectv::updatemu(float4* mu, float4* h2, float4* psi, float rho, hipStream_t s)
{
    updatemu_ker<<<GS3d4, BS3d, 0, s>>>(mu, h2, psi, rho, n+1, (m+1)*(nzp+1));    
}

void rectv::solver_admm(float *ft0, float *ftn0, float *h10, float4 *h20, float *g0, float4 *psi0, float4 *mu0, int iz, int titer, int igpu, hipStream_t s)
{
    float rho = 0.5;

    cg(ft0, ftn0, h10, h20, g0, psi0, mu0, rho, iz, titer, igpu, s);         
    solve_reg(psi0, h20, mu0, lambda0, rho, s);
    updatemu(mu0, h20, psi0, rho, s);
    // hipDeviceSynchronize();
    // double norm=0;
    // for( int id=0;id< (n + 1)*(n + 1) * (m + 1) * (nzp);id++) 
    //     norm+=sqrt(h20[id].x * h20[id].x + h20[id].y * h20[id].y + h20[id].z * h20[id].z + h20[id].w * h20[id].w);

    // printf(" %f\n",lambda0*norm);

}
