#include "hip/hip_runtime.h"
#include "radonusfft.cuh"
#include "kernels_radonusfft.cuh"
#include <stdio.h>

radonusfft::radonusfft(size_t n_, size_t ntheta_, size_t nz_, float center_)
{
	n = n_;
	ntheta = ntheta_;
	nz = nz_;
	center = center_;
	float eps = 1e-3; // accuracy of USFFT
	mu = -log(eps) / (2 * n * n);
	m = ceil(2 * n * 1 / PI * sqrt(-mu * log(eps) + (mu * n) * (mu * n) / 4)); // interpolation radius according to accuracy
	hipMalloc((void **)&f, n * n * nz * sizeof(float2));
	hipMalloc((void **)&g, n * ntheta * nz * sizeof(float2));
	hipMalloc((void **)&fde, (2 * n + 2 * m) * (2 * n + 2 * m) * nz * sizeof(float2));
	hipMalloc((void **)&x, n * ntheta * sizeof(float));
	hipMalloc((void **)&y, n * ntheta * sizeof(float));
	hipMalloc((void **)&theta, ntheta * sizeof(float));

	int ffts[2];
	int idist;
	int odist;
	int inembed[2];
	int onembed[2];
	//fft 2d
	ffts[0] = 2 * n;
	ffts[1] = 2 * n;
	idist = (2 * n + 2 * m) * (2 * n + 2 * m);
	odist = (2 * n + 2 * m) * (2 * n + 2 * m);
	inembed[0] = 2 * n + 2 * m;
	inembed[1] = 2 * n + 2 * m;
	onembed[0] = 2 * n + 2 * m;
	onembed[1] = 2 * n + 2 * m;
	hipfftPlanMany(&plan2d, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, nz);

	//fft 1d
	ffts[0] = n;
	idist = n;
	odist = n;
	inembed[0] = n;
	onembed[0] = n;
	hipfftPlanMany(&plan1d, 1, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, ntheta * nz);
	
  	hipMalloc((void **)&shiftfwd, n * sizeof(float2));
  	hipMalloc((void **)&shiftadj, n * sizeof(float2));
  	// compute shifts with respect to the rotation center
  	takeshift <<<ceil(n / 1024.0), 1024>>> (shiftfwd, -(center - n / 2.0), n);
  	takeshift <<<ceil(n / 1024.0), 1024>>> (shiftadj, (center - n / 2.0), n);

	BS2d = dim3(32, 32);
	BS3d = dim3(32, 32, 1);

	GS2d0 = dim3(ceil(n / (float)BS2d.x), ceil(ntheta / (float)BS2d.y));
	GS3d0 = dim3(ceil(n / (float)BS3d.x), ceil(n / (float)BS3d.y), ceil(nz / (float)BS3d.z));
	GS3d1 = dim3(ceil(2 * n / (float)BS3d.x), ceil(2 * n / (float)BS3d.y), ceil(nz / (float)BS3d.z));
	GS3d2 = dim3(ceil((2 * n + 2 * m) / (float)BS3d.x), ceil((2 * n + 2 * m) / (float)BS3d.y), ceil(nz / (float)BS3d.z));
	GS3d3 = dim3(ceil(n / (float)BS3d.x), ceil(ntheta / (float)BS3d.y), ceil(nz / (float)BS3d.z));	
}

radonusfft::~radonusfft()
{
	hipFree(f);
	hipFree(g);
	hipFree(fde);
	hipFree(x);
	hipFree(y);
	hipFree(shiftfwd);
	hipFree(shiftadj);
	hipfftDestroy(plan2d);
	hipfftDestroy(plan1d);
}

void radonusfft::fwdR(float2 *g_, float2 *f_, float *theta_, hipStream_t s)
{	
	//NOTE: SIZE(g) = [nz,ntheta,n]
	hipMemcpyAsync(f, f_, n * n * nz * sizeof(float2), hipMemcpyDefault, s);
	hipMemcpyAsync(theta, theta_, ntheta * sizeof(float), hipMemcpyDefault, s);
	hipMemsetAsync(fde, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * nz * sizeof(float2), s);

	takexy<<<GS2d0, BS2d, 0, s>>>(x, y, theta, n, ntheta);
	divphi<<<GS3d0, BS3d, 0, s>>>(fde, f, mu, m, n, nz, TOMO_FWD);
	
	fftshift<<<GS3d2, BS3d, 0, s>>>(fde, 2 * n + 2 * m,2 * n + 2 * m, nz, 1);
	hipfftSetStream(plan2d, s);
	hipfftExecC2C(plan2d, (hipfftComplex *)&fde[m + m * (2 * n + 2 * m)], (hipfftComplex *)&fde[m + m * (2 * n + 2 * m)], HIPFFT_FORWARD);
	fftshift<<<GS3d2, BS3d, 0, s>>>(fde, 2 * n + 2 * m, 2 * n + 2 * m, nz, 1);
	
	wrap<<<GS3d2, BS3d, 0, s>>>(fde, n, nz, m, TOMO_FWD);
	gather<<<GS3d3, BS3d, 0, s>>>(g, fde, x, y, m, mu, n, ntheta, nz, TOMO_FWD);	
	
	// shift with respect to given center
  	shift <<<GS3d3, BS3d, 0, s>>> (g, shiftfwd, n, ntheta, nz);	  
	fftshift<<<GS3d3, BS3d, 0, s>>>(g, n, ntheta, nz, 0);
	hipfftSetStream(plan1d, s);
	hipfftExecC2C(plan1d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_BACKWARD);
	fftshift<<<GS3d3, BS3d, 0, s>>>(g, n, ntheta, nz, 0);
	
	mulr<<<GS3d3, BS3d, 0, s>>>(g, 1.0f / sqrt(n * ntheta), n, ntheta, nz);
	hipMemcpyAsync(g_, g, n * ntheta * nz * sizeof(float2), hipMemcpyDefault, s);
}

void radonusfft::adjR(float2 *f_, float2 *g_, float *theta_, bool filter, hipStream_t s)
{
	//NOTE: SIZE(g) = [nz,ntheta,n]
	hipMemcpyAsync(g, g_, n * ntheta * nz * sizeof(float2), hipMemcpyDefault, s);
	hipMemcpyAsync(theta, theta_, ntheta * sizeof(float), hipMemcpyDefault, s);
	hipMemsetAsync(fde, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * nz * sizeof(float2), s);

	takexy<<<GS2d0, BS2d, 0, s>>>(x, y, theta, n, ntheta);

	fftshift<<<GS3d3, BS3d, 0, s>>>(g, n, ntheta, nz, 0 );
	hipfftSetStream(plan1d, s);
	hipfftExecC2C(plan1d, (hipfftComplex *)g, (hipfftComplex *)g, HIPFFT_FORWARD);
	fftshift<<<GS3d3, BS3d, 0, s>>>(g, n, ntheta, nz, 0);
	
	// shift with respect to given center	
	shift <<<GS3d3, BS3d, 0 , s>>> (g, shiftadj, n, ntheta, nz);
	gather<<<GS3d3, BS3d, 0, s>>>(g, fde, x, y, m, mu, n, ntheta, nz, TOMO_ADJ);	
	wrap<<<GS3d2, BS3d, 0, s>>>(fde, n, nz, m, TOMO_ADJ);
	
	fftshift<<<GS3d2, BS3d, 0, s>>>(fde, 2 * n + 2 * m, 2 * n + 2 * m, nz, 1);
	
	hipfftSetStream(plan2d, s);
	hipfftExecC2C(plan2d, (hipfftComplex *)&fde[m + m * (2 * n + 2 * m)], (hipfftComplex *)&fde[m + m * (2 * n + 2 * m)], HIPFFT_BACKWARD);
	fftshift<<<GS3d2, BS3d, 0, s>>>(fde, 2 * n + 2 * m, 2 * n + 2 * m, nz, 1);
	divphi<<<GS3d0, BS3d, 0, s>>>(fde, f, mu, m, n, nz, TOMO_ADJ);
	
	mulr<<<GS3d0, BS3d, 0, s>>>(f, 1.0f / sqrt(n * ntheta), n, n, nz);

	hipMemcpyAsync(f_, f, n * n * nz * sizeof(float2), hipMemcpyDefault, s);
}
